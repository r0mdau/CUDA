#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include "common\cpu_bitmap.h"
#include "common\book.h"

#include "hip/hip_runtime.h"
#include ""

#include "sphere.h"

#define DIM 512
#define NUM_SPHERES 10
texture<unsigned char, 2> texConstSrc;

struct Sphere{
	int r, g ,b;
	float rayon;
	float x, y, z;
	void init(int or, int og, int ob, float ox, float oy, float oz, float orayon){
		r = or; g = og; b = ob;
		x = ox; y = oy; z = oz;
		rayon = orayon;
	}
	__device__ float touche(float ox, float oy){
		float distance = sqrt(pow((ox - x), 2) + pow((oy - y), 2));
		if(distance < rayon){
			return distance;
		}else return 0;
	}
};

__global__ void kernel (Sphere * tab, unsigned char *ptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float ox = x;
	float oy = y;

	ptr[offset * 4 + 0] = 255;
	ptr[offset * 4 + 1] = 255;
	ptr[offset * 4 + 2] = 255;
	ptr[offset * 4 + 3] = 255;

	for(int i = 0; i < NUM_SPHERES; i++){
		float distance = tab[i].touche(ox, oy);
		if(distance){
			float attenuation = 1 - (distance / tab[i].rayon / 1.2);
			ptr[offset * 4 + 0] = tab[i].r * attenuation;
			ptr[offset * 4 + 1] = tab[i].g * attenuation;
			ptr[offset * 4 + 2] = tab[i].b * attenuation;
			ptr[offset * 4 + 3] = 255;
		}
	}
}
__global__ void kernelRotate (unsigned char *ptr, float deg) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	tex2D(texConstSrc, x, y);
}

__device__ float readPixVal( unsigned char* ImgSrc,int ImgWidth,int x,int y)
{
    return (float)ImgSrc[y*ImgWidth+x];
}

__device__ void putPixVal( unsigned char* ImgSrc,int ImgWidth,int x,int y, float floatVal)
{
    ImgSrc[y*ImgWidth+x] = floatVal;
}

void mainSphere (void) {
	srand(time(NULL));
	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;
	Sphere *scene;

	Sphere tab[NUM_SPHERES];
	for(int i = 0; i < NUM_SPHERES; i++){
		tab[i].init(rand() % 256, rand() % 256, rand() % 256, DIM/2, rand() % DIM, rand() % DIM, 30);
	}

	hipMalloc( (void**)&scene, sizeof(Sphere) * NUM_SPHERES);
	hipMemcpy(scene, tab, sizeof(Sphere) * NUM_SPHERES, hipMemcpyHostToDevice);

	hipMalloc( (void**)&dev_bitmap, bitmap.image_size());

	dim3 grid(DIM/16, DIM/16);
	dim3 threads(16, 16);
	
	kernel<<<grid, threads>>>(scene, dev_bitmap);

	hipMalloc((void**)&texConstSrc,  DIM * DIM * 4);
	hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
	hipBindTexture2D( NULL, texConstSrc, dev_bitmap, desc, DIM, DIM, DIM * DIM * 4 );

	kernelRotate<<<grid, threads>>>(dev_bitmap, 45);

	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);	

	bitmap.display_and_exit();

	hipFree(dev_bitmap);
	hipFree(scene);
}